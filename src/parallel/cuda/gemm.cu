/**
 * @file gemm.cu
 * @author Yangyang Zhu (1929772352@qq.com)
 * @version 0.1
 * @date 2024-07-31
 * 
 * @copyright Copyright (c) 2024
 * gpu accelerate gemm
 * reference: https://dlsyscourse.org/slides/12-gpu-acceleration.pdf
 */

#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// #define TILE_SIZE 2
#define TILE_SIZE 16

#define CUDA_CHECK(call)                                                    \
{                                                                           \
    const hipError_t error = call;                                         \
    if (error != hipSuccess)                                               \
    {                                                                       \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", ";      \
        std::cerr << "code: " << error << ", reason: " << hipGetErrorString(error) << std::endl; \
        exit(1);                                                            \
    }                                                                       \
}

#define CUBLAS_CHECK(call)                                                  \
{                                                                           \
    const hipblasStatus_t status = call;                                     \
    if (status != HIPBLAS_STATUS_SUCCESS)                                    \
    {                                                                       \
        std::cerr << "CUBLAS Error: " << __FILE__ << ":" << __LINE__ << ", "; \
        std::cerr << "status: " << status << std::endl;                     \
        exit(1);                                                            \
    }                                                                       \
}

__global__ void gemm_kernel_naive(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float value = 0.0f;
        for (int e = 0; e < K; ++e) {
            value += A[row * K + e] * B[e * N + col];
        }
        C[row * N + col] = value;
    }
}

void gemm_cuda_naive(float* A, float* B, float* C, int M, int N, int K) {
    float *d_A, *d_B, *d_C;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    CUDA_CHECK(hipMalloc((void**)&d_A, size_A));
    CUDA_CHECK(hipMalloc((void**)&d_B, size_B));
    CUDA_CHECK(hipMalloc((void**)&d_C, size_C));

    // CUDA_CHECK(cudaMallocManaged((void**)&d_A, size_A));
    // CUDA_CHECK(cudaMallocManaged((void**)&d_B, size_B));
    // CUDA_CHECK(cudaMallocManaged((void**)&d_C, size_C));

    CUDA_CHECK(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));

    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
    
    gemm_kernel_naive<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, K);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
}

void gemm_cuda_cublas(float* A, float* B, float* C, int M, int N, int K) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    float *d_A, *d_B, *d_C;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    CUDA_CHECK(hipMalloc((void**)&d_A, size_A));
    CUDA_CHECK(hipMalloc((void**)&d_B, size_B));
    CUDA_CHECK(hipMalloc((void**)&d_C, size_C));

    CUDA_CHECK(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));

    float alpha = 1.0f;
    float beta = 0.0f;

    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));

    CUDA_CHECK(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

/************************************** reg tile *********************************************/
// #define reg_tile_size 1
// #define reg_tile_size 2
#define reg_tile_size 4

// __device__ void get_slice() {
//     for (int i = 0; i < reg_tile_size; ++i) {
//         array[i] = 
//     }
// }
// 
// __device__ void set_slice() {
// 
// }

__global__ void gemm_kernel_reg_tile(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float a[reg_tile_size];
    float b[reg_tile_size];
    float c[reg_tile_size][reg_tile_size] = {0};

    // use out prod to compute gemm
    for (int k = 0; k < K; ++k) {

        // get a
        for (int i = 0; i < reg_tile_size; ++i) {
            a[i] = A[(row * reg_tile_size + i) * K + k];
        }
        // get b
        for (int j = 0; j < reg_tile_size; ++j) {
            b[j] = B[k * N + (col * reg_tile_size + j)];
        }

        // out prod of matmul
        for (int y = 0; y < reg_tile_size; ++y) {
            for (int x = 0; x < reg_tile_size; ++x) {
                c[y][x] += a[y] * b[x];
            }
        }

    }

    // set C
    for (int y = 0; y < reg_tile_size; ++y) {
        for (int x = 0; x < reg_tile_size; ++x) {
            C[(row * reg_tile_size + y) * N + (col * reg_tile_size + x)] = c[y][x];
            // printf c[y][x]
            // printf("%f\n", c[y][x]);
        }
    }
}

void gemm_cuda_reg_tile(float* A, float* B, float* C, int M, int N, int K) {
    float *d_A, *d_B, *d_C;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    CUDA_CHECK(hipMalloc((void**)&d_A, size_A));
    CUDA_CHECK(hipMalloc((void**)&d_B, size_B));
    CUDA_CHECK(hipMalloc((void**)&d_C, size_C));

    // CUDA_CHECK(cudaMallocManaged((void**)&d_A, size_A));
    // CUDA_CHECK(cudaMallocManaged((void**)&d_B, size_B));
    // CUDA_CHECK(cudaMallocManaged((void**)&d_C, size_C));

    CUDA_CHECK(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));

    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((N + TILE_SIZE - 1) / TILE_SIZE / reg_tile_size, (M + TILE_SIZE - 1) / TILE_SIZE / reg_tile_size);
    
    gemm_kernel_reg_tile<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, K);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
}
