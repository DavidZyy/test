#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", \
                   __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

__global__ void computeKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] + 1.0f;
    }
}

void kernel_time(float *buffer, int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;
    int times = 10;

    CUDA_CHECK(hipEventRecord(start));
    // launch times kernel(for example, 100 launches overlap one memcpy)
    for (int i = 0; i < times; i++) {
        computeKernel<<<gridSize, blockSize>>>(buffer, size);
    }
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for kernel execution: %.2f ms\n", milliseconds);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

void copy_time(float *dst, float *src,  int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    CUDA_CHECK(hipEventRecord(start));

    CUDA_CHECK(hipMemcpy(dst, src, size* sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for memory copy: %.2f ms\n", milliseconds);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

void copy_time_async(float *dst, float *src,  int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    CUDA_CHECK(hipEventRecord(start));

    CUDA_CHECK(hipMemcpyAsync(dst, src, size* sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for memory copy async: %.2f ms\n", milliseconds);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

void runSameStream(float *buffer, float *dst, float *src, int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;
    int times = 10;

    CUDA_CHECK(hipEventRecord(start));

    CUDA_CHECK(hipMemcpy(dst, src, size* sizeof(float), hipMemcpyHostToDevice));
    for (int i = 0; i < times; i++) {
        computeKernel<<<gridSize, blockSize>>>(buffer, size);
    }
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for same stream: %.2f ms\n", milliseconds);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

void runDifferentStreams(float *buffer, float *dst, float *src, int size, int gridSize, int blockSize) {
    hipStream_t stream1, stream2;
    hipEvent_t start, stop, kernelDone;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    // CUDA_CHECK(cudaEventCreate(&kernelDone));
    float milliseconds = 0;
    int times = 10;

    CUDA_CHECK(hipEventRecord(start, stream1));

    CUDA_CHECK(hipMemcpyAsync(dst, src, size * sizeof(float), hipMemcpyHostToDevice, stream2));
    for (int i = 0; i < times; i++) {
        computeKernel<<<gridSize, blockSize,0, stream1>>>(buffer, size);
    }

    CUDA_CHECK(hipEventRecord(stop, stream2));
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));

    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for different streams: %.2f ms\n", milliseconds);

    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

int main() {
    const int SIZE = 1 << 26;
    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float *h_buffer = (float*)malloc(SIZE * sizeof(float));
    for (int i = 0; i < SIZE; i++) {
        h_buffer[i] = (float)1.0f;
    }
    
    float* d_buffer1, *d_buffer2;
    CUDA_CHECK(hipMalloc((void**)&d_buffer1, SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_buffer2, SIZE * sizeof(float)));

    // copy h_buffer to d_buffer1
    CUDA_CHECK(hipMemcpy(d_buffer1, h_buffer, SIZE * sizeof(float), hipMemcpyHostToDevice));

    // warm up
    for (int i = 0; i < 10; i++) {
        computeKernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_buffer1, SIZE);
    }

    runDifferentStreams(d_buffer1, d_buffer2, h_buffer, SIZE, GRID_SIZE, BLOCK_SIZE);
    runSameStream(d_buffer1, d_buffer2, h_buffer, SIZE, GRID_SIZE, BLOCK_SIZE);
    copy_time(d_buffer2, h_buffer, SIZE, GRID_SIZE, BLOCK_SIZE);
    // copy_time_async(d_buffer2, h_buffer, SIZE, GRID_SIZE, BLOCK_SIZE);
    kernel_time(d_buffer1, SIZE, GRID_SIZE, BLOCK_SIZE);
}
