#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", \
                   __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


__global__ void computeKernel(int* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] + 1.0f;
    }
}

int times = 10;

__global__ void compareKernel(int* data, int times, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        if (data[idx] != times) {
            printf("Mismatch at index %d: %d != %d\n", idx, data[idx], times);
        }
    }
}

void kernel_time(int *buffer, int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    CUDA_CHECK(hipEventRecord(start));

    for (int i = 0; i < times; i++) {
        computeKernel<<<gridSize, blockSize>>>(buffer, size);
    }

    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(cudaDeviceSynchronize()); // no need

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for kernel execution: %.2f ms\n", milliseconds);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    compareKernel<<<gridSize, blockSize>>>(buffer, times, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

void copy_time(int *dst, int *src,  int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    CUDA_CHECK(hipEventRecord(start));

    CUDA_CHECK(hipMemcpy(dst, src, size* sizeof(int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(cudaDeviceSynchronize()); no need 
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for memory copy: %.2f ms\n", milliseconds);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    compareKernel<<<gridSize, blockSize>>>(dst, times, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize()); // Wait for kernel to finish, or you might not see the output in compareKernel
}

void runSameStream(int *buffer, int *dst, int *src, int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;
    int times = 10;

    CUDA_CHECK(hipEventRecord(start));

    CUDA_CHECK(hipMemcpy(dst, src, size* sizeof(int), hipMemcpyHostToDevice));
    for (int i = 0; i < times; i++) {
        computeKernel<<<gridSize, blockSize>>>(buffer, size);
    }
    CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(cudaDeviceSynchronize()); // no need stop will wait kernel to finish

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for same stream: %.2f ms\n", milliseconds);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    compareKernel<<<gridSize, blockSize>>>(buffer, times, size);
    compareKernel<<<gridSize, blockSize>>>(dst, times, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

void runDifferentStreams(int *buffer, int *dst, int *src, int size, int gridSize, int blockSize) {
    hipStream_t stream1, stream2;
    hipEvent_t start1, stop1, start2, stop2;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));
    CUDA_CHECK(hipEventCreate(&start1));
    CUDA_CHECK(hipEventCreate(&stop1));
    CUDA_CHECK(hipEventCreate(&start2));
    CUDA_CHECK(hipEventCreate(&stop2));
    float milliseconds = 0;
    int times = 10;

    CUDA_CHECK(hipEventRecord(start1, stream1));
    CUDA_CHECK(hipEventRecord(start2, stream2));

    // CUDA_CHECK(cudaMemcpyAsync(dst, src, size * sizeof(int), cudaMemcpyHostToDevice, stream2));  // put here have no overlap ...
    for (int i = 0; i < times; i++) {
        computeKernel<<<gridSize, blockSize,0, stream1>>>(buffer, size);
    }
    CUDA_CHECK(hipMemcpyAsync(dst, src, size * sizeof(int), hipMemcpyHostToDevice, stream2)); // put here have overlap ...

    // CUDA_CHECK(cudaDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(stop1, stream1));
    CUDA_CHECK(hipEventRecord(stop2, stream2));
    // CUDA_CHECK(cudaStreamSynchronize(stream1));
    // CUDA_CHECK(cudaStreamSynchronize(stream2));
    CUDA_CHECK(hipDeviceSynchronize());

    // find the max time of start1 - stop1 and start2 - stop2, start1 - stop2 and start2 - stop1
    // ...
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start1, stop2));
    printf("Time for different streams: %.2f ms\n", milliseconds);

    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));
    CUDA_CHECK(hipEventDestroy(start1));
    CUDA_CHECK(hipEventDestroy(stop1));
    CUDA_CHECK(hipEventDestroy(start2));
    CUDA_CHECK(hipEventDestroy(stop2));

    compareKernel<<<gridSize, blockSize>>>(buffer, times, size);
    compareKernel<<<gridSize, blockSize>>>(dst, times, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

int main() {
    const int SIZE = 1 << 26;
    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int *h_buffer, *d_buffer1, *d_buffer2;

    h_buffer = (int*)malloc(SIZE * sizeof(int));
    CUDA_CHECK(hipMalloc((void**)&d_buffer1, SIZE * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_buffer2, SIZE * sizeof(int)));

    for (int i = 0; i < SIZE; i++) {
        h_buffer[i] = times;
    }

    CUDA_CHECK(hipMemset(d_buffer1, 0, SIZE * sizeof(int)));
    CUDA_CHECK(hipMemset(d_buffer2, 0, SIZE * sizeof(int)));

    runDifferentStreams(d_buffer1, d_buffer2, h_buffer, SIZE, GRID_SIZE, BLOCK_SIZE);
    CUDA_CHECK(hipMemset(d_buffer1, 0, SIZE * sizeof(int)));
    CUDA_CHECK(hipMemset(d_buffer2, 0, SIZE * sizeof(int)));

    kernel_time(d_buffer1, SIZE, GRID_SIZE, BLOCK_SIZE);
    copy_time(d_buffer2, h_buffer, SIZE, GRID_SIZE, BLOCK_SIZE);
    CUDA_CHECK(hipMemset(d_buffer1, 0, SIZE * sizeof(int)));
    CUDA_CHECK(hipMemset(d_buffer2, 0, SIZE * sizeof(int)));

    runSameStream(d_buffer1, d_buffer2, h_buffer, SIZE, GRID_SIZE, BLOCK_SIZE);
    CUDA_CHECK(hipMemset(d_buffer1, 0, SIZE * sizeof(int)));
    CUDA_CHECK(hipMemset(d_buffer2, 0, SIZE * sizeof(int)));
}
