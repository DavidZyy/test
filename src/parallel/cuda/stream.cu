// overlap gpu data transfer and kernel execution

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", \
                   __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

__global__ void computeKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] * 2.0f;
    }
}

void runSameStream(float* h_input, float* h_output, float* d_input, int size, int gridSize, int blockSize) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    CUDA_CHECK(hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice));
    
    printf("Testing kernel and memcpy in same stream...\n");
    CUDA_CHECK(hipEventRecord(start));
    computeKernel<<<gridSize, blockSize>>>(d_input, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipMemcpy(h_output, d_input, size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for same stream: %.2f ms\n", milliseconds);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

void runDifferentStreams(float* h_input, float* h_output, float* d_input, int size, int gridSize, int blockSize) {
    hipStream_t stream1, stream2;
    hipEvent_t start, stop, kernelDone;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventCreate(&kernelDone));
    float milliseconds = 0;

    CUDA_CHECK(hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice));
    
    printf("\nTesting kernel and memcpy in different streams...\n");
    CUDA_CHECK(hipEventRecord(start, stream1));
    
    computeKernel<<<gridSize, blockSize, 0, stream1>>>(d_input, size);
    // CUDA_CHECK(cudaEventRecord(kernelDone, stream1));  // 标记kernel完成
    
    // 确保memcpy在kernel完成后执行
    // CUDA_CHECK(cudaStreamWaitEvent(stream2, kernelDone, 0));
    CUDA_CHECK(hipMemcpyAsync(h_output, d_input, size * sizeof(float), hipMemcpyDeviceToHost, stream2));
    
    CUDA_CHECK(hipEventRecord(stop, stream2));
    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Time for different streams: %.2f ms\n", milliseconds);

    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipEventDestroy(kernelDone));
}

int main() {
    const int SIZE = 1 << 30;
    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float *h_input = (float*)malloc(SIZE * sizeof(float));
    float *h_output = (float*)malloc(SIZE * sizeof(float));
    for (int i = 0; i < SIZE; i++) {
        h_input[i] = (float)i;
    }

    float *d_input;
    CUDA_CHECK(hipMalloc(&d_input, SIZE * sizeof(float)));

    // 预热GPU，避免初始化开销影响测量
    computeKernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_input, SIZE);
    CUDA_CHECK(hipDeviceSynchronize());

    // 运行测试
    runSameStream(h_input, h_output, d_input, SIZE, GRID_SIZE, BLOCK_SIZE);
    runDifferentStreams(h_input, h_output, d_input, SIZE, GRID_SIZE, BLOCK_SIZE);
    
    // 验证结果
    bool correct = true;
    for (int i = 0; i < SIZE; i++) {
        if (h_output[i] != h_input[i] * 2.0f) {
            correct = false;
            break;
        }
    }
    printf("Same stream verification: %s\n", correct ? "PASS" : "FAIL");

    runSameStream(h_input, h_output, d_input, SIZE, GRID_SIZE, BLOCK_SIZE);
    runDifferentStreams(h_input, h_output, d_input, SIZE, GRID_SIZE, BLOCK_SIZE);
    
    // 验证结果
    correct = true;
    for (int i = 0; i < SIZE; i++) {
        if (h_output[i] != h_input[i] * 2.0f) {
            correct = false;
            break;
        }
    }
    printf("Different streams verification: %s\n", correct ? "PASS" : "FAIL");

    // 清理
    CUDA_CHECK(hipFree(d_input));
    free(h_input);
    free(h_output);

    return 0;
}
